#include "hip/hip_runtime.h"
#include "hsv_gpu.hpp"

void __global__ hsv_filter(uchar *dev_src, uchar *dev_hsrc, int cols,
			   int hue1, int hue2, int saturation1, int saturation2, int value1, int value2)
{
	int tid = blockIdx.x;
	int c;

	for(c=0;c<cols;c++)
	{
		if(dev_hsrc[cols*3*tid + 3*c + 0] < hue1 || dev_hsrc[cols*3*tid + 3*c + 0] > hue2 ||
		   dev_hsrc[cols*3*tid + 3*c + 1] < saturation1 || dev_hsrc[cols*3*tid + 3*c + 1] > saturation2 ||
		   dev_hsrc[cols*3*tid + 3*c + 2] < value1 || dev_hsrc[cols*3*tid + 3*c + 1] > value2)
		{
			dev_src[cols*3*tid + 3*c + 0] = 0;
			dev_src[cols*3*tid + 3*c + 1] = 0;
			dev_src[cols*3*tid + 3*c + 2] = 0;
		}
	}
}

void __global__ h_filter(uchar *dev_src, uchar *dev_hsrc, int cols, int hue1, int hue2)
{
	int tid = blockIdx.x;
	int c;

	for(c=0;c<cols;c++)
	{
		if(dev_hsrc[cols*3*tid + 3*c + 0] < hue1 || dev_hsrc[cols*3*tid + 3*c + 0] > hue2)
		{
			dev_src[cols*3*tid + 3*c + 0] = 0;
			dev_src[cols*3*tid + 3*c + 1] = 0;
			dev_src[cols*3*tid + 3*c + 2] = 0;
		}
	}
}

void __global__ s_filter(uchar *dev_src, uchar *dev_hsrc, int cols, int saturation1, int saturation2)
{
	int tid = blockIdx.x;
	int c;

	for(c=0;c<cols;c++)
	{
		if(dev_hsrc[cols*3*tid + 3*c + 1] < saturation1 || dev_hsrc[cols*3*tid + 3*c + 1] > saturation2)
		{
			dev_src[cols*3*tid + 3*c + 0] = 0;
			dev_src[cols*3*tid + 3*c + 1] = 0;
			dev_src[cols*3*tid + 3*c + 2] = 0;
		}
	}
}

void __global__ v_filter(uchar *dev_src, uchar *dev_hsrc, int cols, int value1, int value2)
{
	int tid = blockIdx.x;
	int c;

	for(c=0;c<cols;c++)
	{
		if(dev_hsrc[cols*3*tid + 3*c + 2] < value1 || dev_hsrc[cols*3*tid + 3*c + 2] > value2)
		{
			dev_src[cols*3*tid + 3*c + 0] = 0;
			dev_src[cols*3*tid + 3*c + 1] = 0;
			dev_src[cols*3*tid + 3*c + 2] = 0;
		}
	}
}

void HSV::setThreshold(double h1, double h2, double s1, double s2, double v1, double v2)
{
	hue1 = h1;
	hue2 = h2;
	saturation1 = s1;
	saturation2 = s2;
	value1 = v1;
	value2 = v2;
}

void HSV::adjustHSV(Mat* src, Mat* dst)
{
	gsrc.upload(*src);
	gpu::cvtColor(gsrc, gdst, CV_RGB2HSV);
	gdst.download(hsrc);

	hipMalloc((void**)&dev_src, src->rows*src->cols*src->channels()*src->elemSize1());
	hipMalloc((void**)&dev_hsrc, hsrc.rows*hsrc.cols*hsrc.channels()*hsrc.elemSize1());
	hipMemcpy(dev_src, src->data, src->rows*src->cols*src->channels()*src->elemSize1(), hipMemcpyHostToDevice);
	hipMemcpy(dev_hsrc, hsrc.data, hsrc.rows*hsrc.cols*hsrc.channels()*hsrc.elemSize1(), hipMemcpyHostToDevice);

	hsv_filter<<<src->rows, 1>>>(dev_src, dev_hsrc, src->cols, hue1, hue2, saturation1, saturation2, value1, value2);

	hipMemcpy(dst->data, dev_src, src->rows*src->cols*src->channels()*src->elemSize1(), hipMemcpyDeviceToHost);
	
	hipFree(dev_src);
	hipFree(dev_hsrc);
	
}

void HSV::adjustH(Mat* src, Mat* dst)
{
	gsrc.upload(*src);
	gpu::cvtColor(gsrc, gdst, CV_RGB2HSV);
	gdst.download(hsrc);

	hipMalloc((void**)&dev_src, src->rows*src->cols*src->channels()*src->elemSize1());
	hipMalloc((void**)&dev_hsrc, hsrc.rows*hsrc.cols*hsrc.channels()*hsrc.elemSize1());
	hipMemcpy(dev_src, src->data, src->rows*src->cols*src->channels()*src->elemSize1(), hipMemcpyHostToDevice);
	hipMemcpy(dev_hsrc, hsrc.data, hsrc.rows*hsrc.cols*hsrc.channels()*hsrc.elemSize1(), hipMemcpyHostToDevice);

	h_filter<<<src->rows, 1>>>(dev_src, dev_hsrc, src->cols, hue1, hue2);

	hipMemcpy(dst->data, dev_src, src->rows*src->cols*src->channels()*src->elemSize1(), hipMemcpyDeviceToHost);
	
	hipFree(dev_src);
	hipFree(dev_hsrc);
}

void HSV::adjustS(Mat* src, Mat* dst)
{
	gsrc.upload(*src);
	gpu::cvtColor(gsrc, gdst, CV_RGB2HSV);
	gdst.download(hsrc);

	hipMalloc((void**)&dev_src, src->rows*src->cols*src->channels()*src->elemSize1());
	hipMalloc((void**)&dev_hsrc, hsrc.rows*hsrc.cols*hsrc.channels()*hsrc.elemSize1());
	hipMemcpy(dev_src, src->data, src->rows*src->cols*src->channels()*src->elemSize1(), hipMemcpyHostToDevice);
	hipMemcpy(dev_hsrc, hsrc.data, hsrc.rows*hsrc.cols*hsrc.channels()*hsrc.elemSize1(), hipMemcpyHostToDevice);

	s_filter<<<src->rows, 1>>>(dev_src, dev_hsrc, src->cols, saturation1, saturation2);

	hipMemcpy(dst->data, dev_src, src->rows*src->cols*src->channels()*src->elemSize1(), hipMemcpyDeviceToHost);
	
	hipFree(dev_src);
	hipFree(dev_hsrc);
}

void HSV::adjustV(Mat* src, Mat* dst)
{
	gsrc.upload(*src);
	gpu::cvtColor(gsrc, gdst, CV_RGB2HSV);
	gdst.download(hsrc);

	hipMalloc((void**)&dev_src, src->rows*src->cols*src->channels()*src->elemSize1());
	hipMalloc((void**)&dev_hsrc, hsrc.rows*hsrc.cols*hsrc.channels()*hsrc.elemSize1());
	hipMemcpy(dev_src, src->data, src->rows*src->cols*src->channels()*src->elemSize1(), hipMemcpyHostToDevice);
	hipMemcpy(dev_hsrc, hsrc.data, hsrc.rows*hsrc.cols*hsrc.channels()*hsrc.elemSize1(), hipMemcpyHostToDevice);

	v_filter<<<src->rows, 1>>>(dev_src, dev_hsrc, src->cols, value1, value2);

	hipMemcpy(dst->data, dev_src, src->rows*src->cols*src->channels()*src->elemSize1(), hipMemcpyDeviceToHost);
	
	hipFree(dev_src);
	hipFree(dev_hsrc);
}
